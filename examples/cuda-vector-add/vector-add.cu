
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addVectors(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) c[idx] = a[idx] + b[idx];
}

int main() {
    const int N = 1<<20; // 2^20, a pleasing power of 2
    size_t size = N * sizeof(float);

    // Host memory
    float *h_a = new float[N];
    float *h_b = new float[N];
    float *h_c = new float[N];

    // Initialize with philosophical constants
    for(int i = 0; i < N; i++) {
        h_a[i] = 1.618033988749895f; // golden ratio
        h_b[i] = 3.141592653589793f; // π
    }

    // Device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verification
    printf("First element: %f\n", h_c[0]); // Should be φ + π

    // Cleanup
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
